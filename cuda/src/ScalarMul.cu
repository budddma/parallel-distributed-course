#include "hip/hip_runtime.h"
#include <ScalarMul.cuh>

/*
 * Calculates scalar multiplication for block
 */
__global__ void ScalarMulBlock(int numElements, float* vector1, float* vector2, float* result) {
    extern __shared__ float sh_data[];
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    int local_tid = threadIdx.x;

    if (tid < numElements) {
        sh_data[local_tid] = vector1[tid] * vector2[tid];
    }
    __syncthreads();

    for (int step = blockDim.x / 2; step >= 1; step >>= 1) {
        if (threadIdx.x < step) {
            sh_data[local_tid] += sh_data[local_tid + step];
            __syncthreads();
        } else {
            break;
        }
    }

    if (threadIdx.x == 0) {
        result[blockIdx.x] = sh_data[0];
    }
}