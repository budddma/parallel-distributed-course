#include "hip/hip_runtime.h"
#include <CosineVector.cuh>
#include <ScalarMulRunner.cuh>
#include <CommonKernels.cuh>
#include <KernelMul.cuh>

float CosineVector(int numElements, float* vector1, float* vector2, int blockSize) {
  int numBlocks = (numElements + blockSize - 1) / blockSize;
  float scalar_mul = ScalarMulTwoReductions(numElements, vector1, vector2, blockSize);

  float* d_vec1 = nullptr;
  float* d_vec1_sq = nullptr;
  float* d_vec2 = nullptr;
  float* d_vec2_sq = nullptr;
  
  hipMalloc(&d_vec1, numElements * sizeof(float));
  hipMalloc(&d_vec1_sq, numElements * sizeof(float));
  hipMalloc(&d_vec2, numElements * sizeof(float));
  hipMalloc(&d_vec2_sq, numElements * sizeof(float));

  hipMemcpy(d_vec1, vector1, numElements * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_vec2, vector2, numElements * sizeof(float), hipMemcpyHostToDevice);

  KernelMul<<<numBlocks, blockSize>>>(numElements, d_vec1, d_vec1, d_vec1_sq);
  KernelMul<<<numBlocks, blockSize>>>(numElements, d_vec2, d_vec2, d_vec2_sq);

  float* vec1_sq = (float*)calloc(numElements, sizeof(float));
  float* vec2_sq = (float*)calloc(numElements, sizeof(float));

  hipMemcpy(vec1_sq, d_vec1_sq, numElements * sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(vec2_sq, d_vec2_sq, numElements * sizeof(float), hipMemcpyDeviceToHost);

  float vec1_norm_sq = Sum(numElements, vec1_sq, blockSize);
  float vec2_norm_sq = Sum(numElements, vec2_sq, blockSize);

  hipFree(d_vec1);
  hipFree(d_vec1_sq);
  hipFree(d_vec2);
  hipFree(d_vec2_sq);
  free(vec1_sq);
  free(vec2_sq);

  if (vec1_norm_sq == 0 || vec2_norm_sq == 0) {
    return 0.0f;
  }
  return scalar_mul / sqrt(vec1_norm_sq) / sqrt(vec2_norm_sq);
}

