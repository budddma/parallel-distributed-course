#include <ScalarMulRunner.cuh>
#include <KernelMul.cuh>
#include <ScalarMul.cuh>
#include <CommonKernels.cuh>


float ScalarMulTwoReductions(int numElements, float* vector1, float* vector2, int blockSize) {
  int numBlocks = (numElements + blockSize - 1) / blockSize;
  float* result = (float*)calloc(blockSize, sizeof(float));

  float* d_vec1 = nullptr;
  float* d_vec2 = nullptr;
  float* d_vec_mul = nullptr;
  float* d_result = nullptr;

  hipMalloc(&d_vec1, numElements * sizeof(float));
  hipMalloc(&d_vec2, numElements * sizeof(float));
  hipMalloc(&d_vec_mul, numElements * sizeof(float));
  hipMalloc(&d_result, numBlocks * sizeof(float));

  hipMemcpy(d_vec1, vector1, numElements * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_vec2, vector2, numElements * sizeof(float), hipMemcpyHostToDevice);

  KernelMul<<<numBlocks, blockSize>>>(numElements, d_vec1, d_vec2, d_vec_mul);

  int remainderSize = numElements;

  while (numBlocks > 1) {
    ReduceSum<<<numBlocks, blockSize, blockSize * sizeof(float)>>>(remainderSize,
                                                                    d_vec_mul,
                                                                    d_result);
    hipMemcpy(d_vec_mul, d_result, numBlocks * sizeof(float), hipMemcpyDeviceToDevice);

    remainderSize = numBlocks;
    numBlocks = (numBlocks + blockSize - 1) / blockSize;
  }

  ReduceSum<<<1, blockSize, blockSize * sizeof(float)>>>(remainderSize, d_vec_mul, d_result);
  float sum = 0.0f;
  hipMemcpy(&sum, d_result, sizeof(float), hipMemcpyDeviceToHost);

  hipFree(d_vec1);
  hipFree(d_vec2);
  hipFree(d_vec_mul);
  hipFree(d_result);
  free(result);

  return sum;
}

float ScalarMulSumPlusReduction(int numElements, float* vector1, float* vector2, int blockSize) {
  int numBlocks = (numElements + blockSize - 1) / blockSize;

  float* d_vec1 = nullptr;
  float* d_vec2 = nullptr;
  float* d_vec_mul = nullptr;
  float* d_result = nullptr;

  hipMalloc(&d_vec1, numElements * sizeof(float));
  hipMalloc(&d_vec2, numElements * sizeof(float));
  hipMalloc(&d_vec_mul, numElements * sizeof(float));
  hipMalloc(&d_result, blockSize * sizeof(float));

  hipMemcpy(d_vec1, vector1, numElements * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_vec2, vector2, numElements * sizeof(float), hipMemcpyHostToDevice);

  KernelMul<<<numBlocks, blockSize>>>(numElements, d_vec1, d_vec2, d_vec_mul);
  SumBlocks<<<1, blockSize>>>(numElements, d_vec_mul, d_result);
  ReduceSum<<<1, blockSize, blockSize * sizeof(float)>>>(blockSize, d_result, d_vec_mul);

  float sum = 0.0f;
  hipMemcpy(&sum, d_vec_mul, sizeof(float), hipMemcpyDeviceToHost);

  hipFree(d_vec1);
  hipFree(d_vec2);
  hipFree(d_vec_mul);
  hipFree(d_result);

  return sum;
}