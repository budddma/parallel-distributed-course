#include "hip/hip_runtime.h"
#include <MatrixMul.cuh>

__global__
void MatrixMul(int heightA, int widthA, int widthB, float *matrixA, float *matrixB, float *matrixResult) {
extern __shared__ float sh_data[];
    float* A_window = sh_data;
    float* B_window = (float*)&A_window[blockDim.x * blockDim.y];

    size_t row = blockIdx.y * blockDim.y + threadIdx.y;
    size_t col = blockIdx.x * blockDim.x + threadIdx.x;

    int widthA_blocks = (widthA + blockDim.x - 1) / blockDim.x;
    float thread_sum = 0.0;

    for (int aBlockIdx = 0; aBlockIdx < widthA_blocks; ++aBlockIdx) {
        size_t curr_ind_a = aBlockIdx * blockDim.x + threadIdx.x;
        size_t a_idx = row * widthA + curr_ind_a;
        int a_window_ind = threadIdx.y * blockDim.x + threadIdx.x;

        if (row < heightA && curr_ind_a < widthA) {
            A_window[a_window_ind] = matrixA[a_idx];
        } else {
            A_window[a_window_ind] = 0.0f;
        }

        size_t curr_ind_b = aBlockIdx * blockDim.x + threadIdx.y;
        size_t b_idx = curr_ind_b * widthB + col;
        int b_window_ind = threadIdx.y * blockDim.x + threadIdx.x;

        if (curr_ind_b < widthA && col < widthB) {
            B_window[b_window_ind] = matrixB[b_idx];
        } else {
            B_window[b_window_ind] = 0.0f;
        }

        __syncthreads();

        for (int k = 0; k < blockDim.y; ++k) {
            thread_sum += A_window[threadIdx.y * blockDim.x + k] *
                          B_window[k * blockDim.x + threadIdx.x];
        }
        
        __syncthreads();
    }

    size_t res_ind = row * widthB + col;
    if (row < heightA && col < widthB) {
        matrixResult[res_ind] = thread_sum;
    }
}