#include "hip/hip_runtime.h"
#include "KernelAdd.cuh"

__global__ void KernelAdd(int numElements, float* x, float* y, float* result) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;

    for (int i = idx; i < numElements; i += stride) {
        result[i] = x[i] + y[i];
    }
}