#include "hip/hip_runtime.h"
#include <KernelMul.cuh>

__global__ void KernelMul(int numElements, float* x, float* y, float* result) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;

    for (int i = idx; i < numElements; i += stride) {
        result[i] = x[i] * y[i];
    }
}