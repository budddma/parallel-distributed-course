#include "hip/hip_runtime.h"
#include <CommonKernels.cuh>
#include <fstream>

void WriteToFile(const char* filename, const int data_size,
                    const int block_size, const double time) {
    std::fstream outfile;
    outfile.open(filename, std::ios::out | std::ios::app);
    outfile << data_size << ' ' << block_size << ' ' << time << '\n';
    outfile.close();
}

void FillData(float* data, int size, float value) {
    for (int i = 0; i < size; ++i) {
        data[i] = value;
    }
}

__global__ void ReduceSum(int numElements, float* input, float* output) {
    extern __shared__ float sh_data[];
    int tid = blockDim.x * blockIdx.x * 2 + threadIdx.x;
    int local_tid = threadIdx.x;

    if (tid + blockDim.x < numElements) {
        sh_data[local_tid] = input[tid] + input[tid + blockDim.x];
    } else if (tid < numElements) {
        sh_data[local_tid] = input[tid];
    } else {
        sh_data[local_tid] = 0;
    }
    __syncthreads();

    for (int step = blockDim.x / 2; step >= 1; step >>= 1) {
        if (threadIdx.x < step) {
            sh_data[local_tid] += sh_data[local_tid + step];
            __syncthreads();
        } else {
            break;
        }
    }

    if (threadIdx.x == 0) {
        output[blockIdx.x] = sh_data[0];
    }
}

__global__ void SumBlocks(int numElements, float* input, float* result) {
    int local_tid = threadIdx.x;
    float thread_sum = 0.0f;

    for (int i = local_tid; i < numElements; i += blockDim.x) {
        thread_sum += input[i];
    }
    
    result[local_tid] = thread_sum;
}

float Sum(int numElements, float* vector, int blockSize) {
    int numBlocks = (numElements + blockSize - 1) / blockSize;
    float* result = (float*)calloc(blockSize, sizeof(float));

    float* d_vec = nullptr;
    float* d_result = nullptr;

    hipMalloc(&d_vec, numElements * sizeof(float));
    hipMalloc(&d_result, numBlocks * sizeof(float));

    hipMemcpy(d_vec, vector, numElements * sizeof(float), hipMemcpyHostToDevice);

    int remainderSize = numElements;

    while (numBlocks > 1) {
        ReduceSum<<<numBlocks, blockSize, blockSize * sizeof(float)>>>(remainderSize,
                                                                       d_vec,
                                                                       d_result);
        hipMemcpy(d_vec, d_result, numBlocks * sizeof(float), hipMemcpyDeviceToDevice);

        remainderSize = numBlocks;
        numBlocks = (numBlocks + blockSize - 1) / blockSize;
    }

    hipMemcpy(result, d_vec, remainderSize * sizeof(float), hipMemcpyDeviceToHost);

    float sum = 0.0;
    for (int i = 0; i < remainderSize; ++i) {
        sum += result[i];
    }

    hipFree(d_vec);
    hipFree(d_result);
    free(result);

    return sum;
}