#include "hip/hip_runtime.h"
#include "KernelAdd.cuh"
#include <iostream>
#include <CommonKernels.cuh>

int main(int argc, char** argv) {
    int blockSize = atoi(argv[1]);
    int N = atoi(argv[2]);
    int numBlocks = std::min(maxBlocks, (N + blockSize - 1) / blockSize);

    size_t size = N * sizeof(float);
    float *x = (float*)calloc(N, sizeof(float));
    float *y = (float*)calloc(N, sizeof(float));
    float *res = (float*)calloc(N, sizeof(float));

    float *d_x, *d_y, *d_res;

    hipMalloc(&d_x, size);
    hipMalloc(&d_y, size);
    hipMalloc(&d_res, size);

    for (int i = 0; i < N; ++i) {
        x[i] = i;
        y[i] = -i;
    }

    hipMemcpy(d_x, x, size, hipMemcpyHostToDevice);
    hipMemcpy(d_y, y, size, hipMemcpyHostToDevice);

    hipEvent_t start;
    hipEvent_t stop;

    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    KernelAdd<<<numBlocks, blockSize>>>(N, d_x, d_y, d_res);

    hipEventRecord(stop);
    hipEventSynchronize(stop);

    hipMemcpy(res, d_res, size, hipMemcpyDeviceToHost);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    
    WriteToFile("data/01-add.txt", N, blockSize, milliseconds);

    hipFree(d_x);
    hipFree(d_y);
    hipFree(d_res);

    free(x);
    free(y);
    free(res);
    return 0;
}