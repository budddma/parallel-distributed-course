#include "hip/hip_runtime.h"
#include <MatrixVectorMul.cuh>
#include <iostream>
#include <CommonKernels.cuh>

int main(int argc, char** argv) {
    int blockSize = atoi(argv[1]);
    int matXSize = atoi(argv[2]);
    int matYSize = atoi(argv[3]);
    
    size_t N = matXSize * matYSize;

    float *mat = (float*)malloc(N * sizeof(float));
    float *vec = (float*)malloc(matXSize * sizeof(float));
    float *res = (float*)malloc(matYSize * sizeof(float));

    FillData(mat, N);
    FillData(vec, matXSize);

    size_t pitch = 0;
    float *d_mat, *d_vec, *d_res;

    hipMallocPitch(&d_mat, &pitch, matXSize * sizeof(float), matYSize);
    hipMallocPitch(&d_vec, &pitch, matXSize * sizeof(float), 1);
    hipMalloc(&d_res, matYSize * sizeof(float));

    hipMemcpy2D(d_mat, pitch, mat, matXSize * sizeof(float), matXSize * sizeof(float), matYSize,
                 hipMemcpyHostToDevice);
    hipMemcpy2D(d_vec, pitch, vec, matXSize * sizeof(float), matXSize * sizeof(float), 1,
                 hipMemcpyHostToDevice);

    hipEvent_t start;
    hipEvent_t stop;

    hipEventCreate(&start);
    hipEventCreate(&stop);

    int numBlocks = std::min(maxBlocks, (matYSize + blockSize - 1) / blockSize);
    hipEventRecord(start);

    MatrixVectorMul<<<numBlocks, blockSize>>>(matYSize, matXSize, d_mat, d_vec, d_res);

    hipEventRecord(stop);
    hipEventSynchronize(stop);

    hipMemcpy(res, d_res, matYSize * sizeof(float), hipMemcpyDeviceToHost);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    WriteToFile("data/04-matrix-vector-mul.txt", matXSize * matYSize, blockSize, milliseconds);

    hipFree(d_mat);
    hipFree(d_vec);
    hipFree(d_res);

    free(mat);
    free(vec);
    free(res);
    return 0;
}