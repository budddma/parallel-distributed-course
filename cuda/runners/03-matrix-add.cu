#include "hip/hip_runtime.h"
#include <KernelMatrixAdd.cuh>
#include <iostream>
#include <CommonKernels.cuh>

int main(int argc, char** argv) {
    int blockSize1D = atoi(argv[1]);
    int matXSize = atoi(argv[2]);
    int matYSize = atoi(argv[3]);

    size_t N = matXSize * matYSize;
    size_t size = N * sizeof(float);
    
    float *x = (float*)malloc(size);
    float *y = (float*)malloc(size);
    float *res = (float*)malloc(size);

    for (int i = 0; i < N; ++i) {
        x[i] = 1.0f;
        y[i] = 2.0f;
    }

    size_t pitch = 0;
    float *d_x, *d_y, *d_res;

    hipMallocPitch(&d_x, &pitch, matXSize * sizeof(float), matYSize);
    hipMallocPitch(&d_y, &pitch, matXSize * sizeof(float), matYSize);
    hipMallocPitch(&d_res, &pitch, matXSize * sizeof(float), matYSize);

    hipMemcpy2D(d_x, pitch, x, matXSize * sizeof(float), matXSize * sizeof(float), matYSize,
                                                                            hipMemcpyHostToDevice);
    hipMemcpy2D(d_y, pitch, y, matXSize * sizeof(float), matXSize * sizeof(float), matYSize,
                                                                            hipMemcpyHostToDevice);

    hipEvent_t start;
    hipEvent_t stop;

    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    int numXBlocks = std::min(maxBlocks, (matXSize + blockSize1D - 1) / blockSize1D);
    int numYBlocks = std::min(maxBlocks, (matYSize + blockSize1D - 1) / blockSize1D);

    dim3 numBlocks(numXBlocks, numYBlocks);
    dim3 blockSize(blockSize1D, blockSize1D);

    KernelMatrixAdd<<<numBlocks, blockSize>>>(matYSize, matXSize, pitch / sizeof(float), d_x, d_y, d_res);

    hipEventRecord(stop);
    hipEventSynchronize(stop);

    hipMemcpy2D(res, matXSize * sizeof(float), d_res, pitch, matXSize * sizeof(float), matYSize,
                    hipMemcpyDeviceToHost);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    WriteToFile("data/03-matrix-add.txt", matXSize * matYSize, blockSize1D, milliseconds);

    hipFree(d_x);
    hipFree(d_y);
    hipFree(d_res);

    free(x);
    free(y);
    free(res);
    return 0;
}