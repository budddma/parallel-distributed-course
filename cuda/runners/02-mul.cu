#include "hip/hip_runtime.h"
#include "KernelMul.cuh"
#include <iostream>
#include <CommonKernels.cuh>

int main(int argc, char** argv) {
	int blockSize = atoi(argv[1]);
    int N = atoi(argv[2]);
    int numBlocks = std::min(maxBlocks, (N + blockSize - 1) / blockSize);

    size_t size = N * sizeof(float);
    float *x = (float*)calloc(N, sizeof(float));
    float *y = (float*)calloc(N, sizeof(float));
    float *res = (float*)calloc(N, sizeof(float));

    float *d_x, *d_y, *d_res;

    hipMalloc(&d_x, size);
    hipMalloc(&d_y, size);
    hipMalloc(&d_res, size);

    for (int i = 0; i < N; ++i) {
        x[i] = 1.0;
        y[i] = 2.0;
    }

    hipMemcpy(d_x, x, size, hipMemcpyHostToDevice);
    hipMemcpy(d_y, y, size, hipMemcpyHostToDevice);

    hipEvent_t start;
    hipEvent_t stop;

    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    KernelMul<<<numBlocks, blockSize>>>(N, d_x, d_y, d_res);

    hipEventRecord(stop);
    hipEventSynchronize(stop);

    hipMemcpy(res, d_res, size, hipMemcpyDeviceToHost);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    WriteToFile("data/02-mul.txt", N, blockSize, milliseconds);

    hipFree(d_x);
    hipFree(d_y);
    hipFree(d_res);

    free(x);
    free(y);
    free(res);
    return 0;
}