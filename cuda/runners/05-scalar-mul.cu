#include <ScalarMulRunner.cuh>
#include <iostream>
#include <CommonKernels.cuh>

int main(int argc, char** argv) {
    int blockSize = atoi(argv[1]);
    int N = atoi(argv[2]);

    size_t size = N * sizeof(float);

    float *x = (float*)malloc(size);

    for (int i = 0; i < N; ++i) {
        x[i] = 1.0f;
    }

    hipEvent_t start;
    hipEvent_t stop;

    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    float first_scalar_mul = ScalarMulSumPlusReduction(N, x, x, blockSize);

    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    WriteToFile("data/05-scalar-1red.txt", N, blockSize, milliseconds);
    hipEventRecord(start);

    float second_scalar_mul = ScalarMulTwoReductions(N, x, x, blockSize);

    hipEventRecord(stop);
    hipEventSynchronize(stop);

    hipEventElapsedTime(&milliseconds, start, stop);

    WriteToFile("data/05-scalar-2red.txt", N, blockSize, milliseconds);

    free(x);
    return 0;
}