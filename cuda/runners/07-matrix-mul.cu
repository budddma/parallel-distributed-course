#include <MatrixMul.cuh>
#include <CommonKernels.cuh>

int main(int argc, char** argv) {
    int blockSize1D = atoi(argv[1]);
    int aHeight = atoi(argv[2]);
    int aWidth = atoi(argv[3]);
    int bWidth = atoi(argv[4]);
    int bHeight = aWidth;

    size_t aN = aHeight * aWidth;
    size_t bN = bHeight * bWidth;
    size_t resN = aHeight * bWidth;

    size_t aSize = aN * sizeof(float);
    size_t bSize = bN * sizeof(float);
    size_t resSize = resN * sizeof(float);

    float *a = (float*)malloc(aSize);
    float *b = (float*)malloc(bSize);
    float *res = (float*)malloc(resSize);

    FillData(a, aN);
    FillData(b, bN, 2.0f);

    size_t a_pitch = 0;
    size_t b_pitch = 0;
    size_t res_pitch = 0;
    float *d_a = nullptr;
    float *d_b = nullptr;
    float *d_res = nullptr;

    hipMallocPitch(&d_a, &a_pitch, aWidth * sizeof(float), aHeight);
    hipMallocPitch(&d_b, &b_pitch, bWidth * sizeof(float), bHeight);
    hipMallocPitch(&d_res, &res_pitch, bWidth * sizeof(float), aHeight);

    hipMemcpy2D(d_a, a_pitch, a, aWidth * sizeof(float), aWidth * sizeof(float), aHeight,
                 hipMemcpyHostToDevice);
    hipMemcpy2D(d_b, b_pitch, b, bWidth * sizeof(float), bWidth * sizeof(float), bHeight,
                 hipMemcpyHostToDevice);

    hipEvent_t start;
    hipEvent_t stop;

    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);

    int numXBlocks = (res_pitch / sizeof(float) + blockSize1D - 1) / blockSize1D;
    int numYBlocks = (aHeight + blockSize1D - 1) / blockSize1D;
    int blockSizeInBytes = blockSize1D * blockSize1D * sizeof(float);

    dim3 numBlocks(numXBlocks, numYBlocks);
    dim3 blockSize(blockSize1D, blockSize1D);

    MatrixMul<<<numBlocks, blockSize, blockSizeInBytes * 2>>>(aHeight,
                                                                a_pitch / sizeof(float),
                                                                b_pitch / sizeof(float),
                                                                d_a,
                                                                d_b,
                                                                d_res);

    hipEventRecord(stop);
    hipEventSynchronize(stop);

    hipMemcpy2D(res, bWidth * sizeof(float), d_res, res_pitch, bWidth * sizeof(float), aHeight,
                 hipMemcpyDeviceToHost);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    WriteToFile("data/07-matrix-mul.txt",
                aWidth * aHeight * bWidth,
                blockSize1D,
                milliseconds);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_res);

    free(a);
    free(b);
    free(res);
    return 0;
}