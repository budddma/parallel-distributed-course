#include <CosineVector.cuh>
#include <iostream>
#include <CommonKernels.cuh>

int main(int argc, char** argv) {
    int blockSize = atoi(argv[1]);
    int N = atoi(argv[2]);

    size_t size = N * sizeof(float);

    float *x = (float*)malloc(size);
    float *y = (float*)malloc(size);

    for (int i = 0; i < N; ++i) {
        x[i] = 1.0f;
        y[i] = -1.0f;
    }

    hipEvent_t start;
    hipEvent_t stop;

    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);

    float cos_angle = CosineVector(N, x, y, blockSize);

    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    WriteToFile("data/06-cosine-vector.txt", N, blockSize, milliseconds);

    free(x);
    free(y);
    return 0;

}
